// lu_cuda.CU

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
using namespace std;

// ——— error‐checking macro ———
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// flat indexing: row-major
__device__ inline int IDX(int row, int col, int n) {
    return row * n + col;
}

// Kernel to compute L[i][k] for i = k..n-1
__global__ void compute_L(const float* __restrict__ a,
                          float* __restrict__ l,
                          const float* __restrict__ u,
                          int k, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i   = tid + k;
    if (i < n) {
        float sum = 0.0f;
        for (int p = 0; p < k; ++p) {
            sum += l[IDX(i,p,n)] * u[IDX(p,k,n)];
        }
        l[IDX(i,k,n)] = a[IDX(i,k,n)] - sum;
    }
}

// Kernel to compute U[k][j] for j = k..n-1
__global__ void compute_U(const float* __restrict__ a,
                          const float* __restrict__ l,
                          float* __restrict__ u,
                          int k, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int j   = tid + k;
    if (j < n) {
        if (j == k) {
            u[IDX(k,k,n)] = 1.0f;
        } else {
            float sum = 0.0f;
            for (int p = 0; p < k; ++p) {
                sum += l[IDX(k,p,n)] * u[IDX(p,j,n)];
            }
            u[IDX(k,j,n)] = (a[IDX(k,j,n)] - sum) / l[IDX(k,k,n)];
        }
    }
}

int main() {
    const char* filename = "Matrix.txt";
    ifstream infile(filename);
    if (!infile.is_open()) {
        cerr << "Error opening " << filename << "\n";
        return 1;
    }

    // --- Read all rows first so we know n ---------------
    vector<string> rows;
    string line;
    while (getline(infile, line, ',')) {
        rows.push_back(line);
    }
    infile.close();
    int n = (int)rows.size();

    // --- Dynamic host storage for A, L, U -------------
    vector<float> h_a(n*n),
                  h_l(n*n, 0.0f),
                  h_u(n*n, 0.0f);

    // Parse each saved row into h_a (flat row-major)
    for (int i = 0; i < n; ++i) {
        stringstream ss(rows[i]);
        for (int j = 0; j < n; ++j) {
            ss >> h_a[i*n + j];
        }
    }

    // Device pointers
    float *d_a, *d_l, *d_u;
    size_t bytes = n * n * sizeof(float);
    CUDA_CHECK(hipMalloc(&d_a, bytes));
    CUDA_CHECK(hipMalloc(&d_l, bytes));
    CUDA_CHECK(hipMalloc(&d_u, bytes));

    // Copy A, zero L and U on device
    CUDA_CHECK(hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_l, 0, bytes));
    CUDA_CHECK(hipMemset(d_u, 0, bytes));

    // --- LU decomposition on GPU ------------------------
    const int TPB = 256;
    for (int k = 0; k < n; ++k) {
        int len    = n - k;
        int blocks = (len + TPB - 1) / TPB;

        compute_L<<<blocks, TPB>>>(d_a, d_l, d_u, k, n);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        compute_U<<<blocks, TPB>>>(d_a, d_l, d_u, k, n);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }

    // Copy back and print
    CUDA_CHECK(hipMemcpy(h_l.data(), d_l, bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_u.data(), d_u, bytes, hipMemcpyDeviceToHost));

    cout << "\nL Decomposition\n";
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j)
            printf("%10.6f ", h_l[i*n + j]);
        cout << "\n";
    }

    cout << "\nU Decomposition\n";
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j)
            printf("%10.6f ", h_u[i*n + j]);
        cout << "\n";
    }

    // Cleanup
    hipFree(d_a);
    hipFree(d_l);
    hipFree(d_u);
    return 0;
}
